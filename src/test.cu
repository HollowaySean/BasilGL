
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_hello() {
    printf("Hello world from GPU!\n");
}

int main(int argc, char** argv) {
    cuda_hello<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
